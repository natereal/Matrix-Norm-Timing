#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>

float max_norm(float *matrix, int m, int n);
float frob_norm(float *matrix, int m, int n);
float one_norm(float *matrix, int m, int n);
float inf_norm(float *matrix, int m, int n);


// #threads
int block_size = 32;	

//================================================================ 
// One global function for each norm, which will
// be called by curly brackets within a function, say,
// 'calculate_norm_on_gpu'


__global__ void add_arrays_gpu(float *matrix, int m, int n){
	// idx is a unique ID for each thread
	//      ( block index * threads ) + thread index
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int k;
	int max = 0;
	if(idx<m) // For each row
		for(k=0; k<n; k++){
			if( fabsf(matrix_d[idx*m + k]) > max ){
				max = fabsf(matrix_d[idx*m + k]);	
			}
		}
	// Store local max somewhere
	
}

//================================================================

double max_norm_gpu(int m, int n ){
	// On device
	float *matrix_d
	hipMalloc ((void **) &matrix_d, sizeof(float)*m*n);
	// Copy data from host memory to device
	hipMemcpy(matrix_d, matrix, sizeof(float)*n*m, hipMemcpyHostToDevice);	

	// Configuring the grid
	dim3 dimBlock(block_size);	// One argument = 1D
	// N/32 blocks, or one extra with an uneven amount
	dim3 dimGrid ( (N/dimBlock.x) + (!(N%dimBlock.x)?0:1) );

	// Matrix for local results
	float max[m];

	// Error Checks

	// Call global function
	compute_maxnorm_gpu<<dimGrid,dimBlock>>(matrix_d,m,n);
	// Run through max[m]
	hipMemcpy(max[idx],max,sizeof(float),hipMemcpyDeviceToHost);

	// return norm
}
int main(int argc, char **argv ){
	
	int c,m,n,i,j;
	int tflag=0,mflag=0,nflag=0;
	m=n=10;

	srand48(123456);

	struct timeval start,end;
	long long time_elasped;

	while((c = getopt(argc,argv,"rtmn")) != -1)
		switch(c){
			case 'r':
				srand48(time(NULL));
				break;
			case 't':
				tflag = 1;
				break;
			case 'm':
				mflag=1;
				break;
			case 'n':
				nflag=1;
				break;
			case '?':
				if(isprint (optopt))
					fprintf(stderr, "Unknown option `-%c.\n",optopt);
				else
					fprintf(stderr,"Unknown option chacracter `\\x%x'.\n",optopt);
				return 1;
			default:
				abort();
		}
	if(mflag)
		m = atoi(argv[optind]);
	if(nflag)
		n = atoi(argv[optind+1]);
	
	printf("m = %d, n = %d\n",m,n);

	//====================================================
 	// Allocate memory 
	// On host
	float *matrix;
	matrix = malloc(n*m*sizeof(float));		
	//====================================================
	
	// Initialise matrix
	for(i=0;i<n*m;i++){
		matrix[i] = drand48();
	}



	float norm;
	// Testing time
	/*
	gettimeofday(&start, NULL);
	sleep(5);
	gettimeofday(&end, NULL);
	printf("%f\n", end.tv_sec - start.tv_sec);
	printf("%f\n", end.tv_usec - start.tv_usec);
	*/
	// Calculating norms
	// Measuring time in microseconds
	gettimeofday(&start, NULL);
	norm = max_norm(matrix,m,n);
	gettimeofday(&end, NULL);
	printf("Max norm %f\n",norm);
	if(tflag){
		time_elasped = (end.tv_sec-start.tv_sec)*1000000 + (end.tv_usec-start.tv_usec);
		printf("\t%lld microseconds \n",time_elasped);
	}
	gettimeofday(&start, NULL);
	norm = frob_norm(matrix,m,n);
	gettimeofday(&end, NULL);
	printf("Frobenius norm %f\n",norm);
	if(tflag){
		time_elasped = (end.tv_sec-start.tv_sec)*1000000 + (end.tv_usec-start.tv_usec);
		printf("\t%lld microseconds \n",time_elasped);
	}
	gettimeofday(&start, NULL);
	norm = one_norm(matrix,m,n);
	gettimeofday(&end, NULL);
	printf("One norm %f\n",norm);
	if(tflag){
		time_elasped = (end.tv_sec-start.tv_sec)*1000000 + (end.tv_usec-start.tv_usec);
		printf("\t%lld microseconds \n",time_elasped);
	}
	gettimeofday(&start, NULL);
	norm = inf_norm(matrix,m,n);
	gettimeofday(&end, NULL);
	printf("Infinity norm %f\n",norm);
	if(tflag){
		time_elasped = (end.tv_sec-start.tv_sec)*1000000 + (end.tv_usec-start.tv_usec);
		printf("\t%lld microseconds \n",time_elasped);
	}

	free(matrix);
	return 0;
}
float max_norm(float *matrix, int m, int n){
	int i, I;
	float max = 0.0;
	for(i=0;i<(n*m);i++){
		if( matrix[i]*matrix[i] > max ){
			max = matrix[i]*matrix[i];
			I = i;
		}
	}
	return fabsf(matrix[I]);
}
float frob_norm(float *matrix, int m, int n){
	int i;
	float sum;
	for(i=0;i<(m*n);i++)
		sum+=matrix[i]*matrix[i];
	return sqrt(sum);	
}
float one_norm(float *matrix, int m, int n){
	// m rows 
	// n columns
	// Access (i.j) entry by matrix[i*n + j]
	float sum,max=0;
	int i,j;
	// Loop over columns
	for(j=0;j<n;j++){
		sum=0;
		// Add all column entries
		for(i=0;i<m;i++){
			sum+=fabsf(matrix[i*n+j]);
		}
		if( sum > max )
			max = sum;
	}
	return max;
}
float inf_norm(float *matrix,int m, int n){
	float sum,max=0;
	int i,j;
	// Loop over rows
	for(i=0;i<m;i++){
		sum=0;
		// Add all entries in row
		for(j=0;j<n;j++){
			sum+=fabsf(matrix[i*n+j]);
		}
		if( sum > max )
			max = sum;
	}
	return max;
}





























